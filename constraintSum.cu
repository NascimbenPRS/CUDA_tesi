#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
//#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>

#include "commonFunctions.h" // generic malloc
#include "sumOptions.h" // options handling


/*
How to compile:
	nvcc "filename" anyoption.cpp
*/


// sum integers dataArr[j], where j belongs to a constraint (array of indexes)
void constraintSum(int *dataArr, int arraySize, int *sumValue, int numCycles, int *constraint, int constraintSize) {
	int tempSum = 0;
	for (int k = 0; k < numCycles; k++) {
		tempSum = 0;
		for (int i = 0; i < constraintSize; i++) {
			tempSum += dataArr[constraint[i]];
		}
	}
	*sumValue = tempSum;
}

// sum integers dataArr[j], where j belongs to a constraint (array of indexes) on GPU
__global__ void constraintSumGPU(int *dataArr, int arraySize, int *sumValues, int numCycles, int numConstraints, int **constraints, int *constraintSizes) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index <= numConstraints) {
		// constraintArr[index] is a valid constraint
		int tempSum = 0;
		for (int k = 0; k < numCycles; k++) {
			tempSum = 0;
			for (int i = 0; i < constraintSizes[index]; i++) {
				tempSum += dataArr[constraints[index][i]];
			}
		}
		sumValues[index] = tempSum;
	}
	else {
		// invalid constraint index
	}
}

// generate random integer in {min,..,max}
int randomIntRange(int min, int max) {
	return min + rand() / (RAND_MAX / (max - min + 1) + 1);
}

// fill a constraint with all-different random integers between 0 and arraySize-1
void fillConstraintRandom(int *constraint, int constraintSize, int arraySize) {
	int temp;
	int k = 0;
	bool isNew = true;
	bool foundNew = false;
	for (int i = 0; i < constraintSize; i++) {
		foundNew = false;
		while (!foundNew) {
			// try new random value
			temp = randomIntRange(0, arraySize - 1);
			isNew = true;
			k = 0;
			while ((k < i) && (isNew)) {
				// compare to previous entries
				if (constraint[k] == temp) {
					isNew = false;
				}
				k++;
			}
			if (isNew) {
				constraint[i] = temp;
				foundNew = true; // allows for-cycle to proceed
			}
		}
	}
}

// get the optimal <<<numOfBlocks, blockSize>>> configuration, given the number of jobs and of multiprocessors available on GPU
void getOptimalGridConfig(int numOfJobs, int numOfMultiProcessors, int maxThreadsPerBlock, int *numOfBlocks, int *blockSize) {
	if (numOfJobs <= numOfMultiProcessors){
		*numOfBlocks = numOfJobs;
		*blockSize = 1;
	}
	else if (numOfJobs <= numOfMultiProcessors * maxThreadsPerBlock){
		*numOfBlocks = numOfMultiProcessors;
		*blockSize = ceil(*numOfBlocks / numOfMultiProcessors);
	}
	else{
		*numOfBlocks = ceil(numOfJobs / maxThreadsPerBlock);
		*blockSize = maxThreadsPerBlock;
	}
}


/*
// initialize array of arrays using multiple threads
__global__ void initializeArraysGPU(int **arrGPU, int arraySize) {
	int index = threadIdx.x; // current thread ID
	for (int i = 0; i < arraySize; i++) {
		arrGPU[index][i] = 1;
	}
}
*/



int main(int argc, char *argv[])
{
	int arraySize = 1 << 20; // 1M integers
	/* VARIABILI INUTILIZZATE
	int usesCache = 1; // 0: don't use cache, 1: use cache (default)
	int cacheLineSize = 64 / sizeof(int); // # integers per cache line on CPU
	int cacheLineSizeGPU = 128 / sizeof(int); // # integers per cache line on GPU
	*/
	int numCycles = 1000; // default # of repetitions on CPU
	int numCyclesGPU = 30; // default # of repetitions on GPU
	int *dataArray, *resultsCPU, *resultsGPU;
	int blockSize = 1; // # of threads per block
	int numBlocks = 1; // # of blocks
	int numThreadsTotal; // # of threads running concurrently (= numThreads * numBlocks)
	int maxBlockSize = 1024;
	int maxNumBlocks = 1024;

	// Constraint variables
	int a = 2 << 10; // # constraints of size 2 (2 variables involved, e.g. x1 + x2 = 0)
	int b = 2 << 10; // # constraints of size 3
	int c = 2 << 10; // # constraints of size in {4..128}
	int numConstraints = a + b + c; // # all constraints
	int *constraintSizes; // constraintSizes[i] == size of constraint i
	int **constraints; // array of constraints
	// set constraint sizes
	genericMalloc((void**)&constraintSizes, numConstraints * sizeof(int));
	for (int i = 0; i < numThreadsTotal; i++) {
		if (i < a) {
			constraintSizes[i] = 2;
		}
		else {
			if (i < a + b) {
				constraintSizes[i] = 3;
			}
			else {
				constraintSizes[i] = randomIntRange(4, 128); // random size 4..128
			}
		}
	}
	// allocate constraints array
	genericMalloc((void**)&constraints, numConstraints * sizeof(int*));
	for (int i = 0; i < numConstraints; i++) {
		genericMalloc((void**)&constraints[i], constraintSizes[i] * sizeof(int));
	}
	// initialize constraints with random values
	for (int i = 0; i < numConstraints; i++) {
		fillConstraintRandom(constraints[i], constraintSizes[i], arraySize);
	}
	// allocate and initialize CPU and GPU results arrays
	genericMalloc((void**)&resultsCPU, numConstraints * sizeof(int));
	genericMalloc((void**)&resultsGPU, sizeof(int));
	for (int i = 0; i < numConstraints; i++) {
		resultsCPU[i] = 0;
		resultsGPU[i] = 0;
	}
	
	
	hipDeviceProp_t prop;
	int device = -1;
	hipGetDevice(&device);
	hipDeviceSynchronize();
	hipGetDeviceProperties(&prop, device);
	int numOfMultiProcessors = prop.multiProcessorCount;

	printf("\nSimulate work on constraints using arrays of integers\n");
	printf("Default options: arraySize= %d integers.\n", arraySize);
	printf("-- number of multiprocessors= %d\n", numOfMultiProcessors);

	/* DA AGGIORNARE CON OPZIONI PER CONSTRAINTS (a,b,c,numConstraints ecc.)
	readOptions(argc, argv, &usesCache, &numCycles, &numCyclesGPU, &arraySize); // read options from command line and update values accordingly
	// don't use --rep
	*/

	// allocate and initialize data array
	genericMalloc((void**)&dataArray, arraySize * sizeof(int));
	for (int i = 0; i < arraySize; i++) {
		dataArray[i] = 1;
	}

	// get optimal GPU grid configuration
	getOptimalGridConfig(numConstraints, numOfMultiProcessors, maxBlockSize, &numBlocks, &blockSize);

	// Time measurement
	int minRunTime = 10; // elapsedTime must be at least minRunTime seconds
	int minNumCyclesCPU = 1, minNumCyclesGPU = 1;
	int elapsedClocks = 0, startClock = 0, endClock = 0;
	double elapsedTimeCPU = 0.f, avgElapsedTimeCPU = 0.f, elapsedTimeGPU = 0.f, avgElapsedTimeGPU = 0.f;

	// Measure CPU execution time
	elapsedTimeCPU = 0.f;
	avgElapsedTimeCPU = 0.f;
	numCycles = minNumCyclesCPU;
	while (elapsedTimeCPU < minRunTime) {// double numCycles until execution takes at least minRunTime
		numCycles *= 2;
		startClock = clock();
		for (int i = 0; i < numConstraints; i++) {
			constraintSum(dataArray, arraySize, &resultsCPU[i], numCycles, constraints[i], constraintSizes[i]);
		}
		endClock = clock();
		elapsedClocks = endClock - startClock;
		elapsedTimeCPU = ((double)(elapsedClocks)) / (CLOCKS_PER_SEC);
	}
	avgElapsedTimeCPU = elapsedTimeCPU / numCycles;
	printf("CPU: Elapsed time= %fs. Average execution time= %fs.\n", elapsedTimeCPU, avgElapsedTimeCPU);
	
	// Measure GPU execution time
	elapsedTimeGPU = 0.f;
	avgElapsedTimeGPU = 0.f;
	numCyclesGPU = minNumCyclesGPU;
	hipMemPrefetchAsync(dataArray, arraySize * sizeof(int), device);
	/*PREFETCH OF CONSTRAINTS AND RESULTSGPU
	*
	*
	*/
	while (elapsedTimeGPU < minRunTime) {// double numCyclesGPU until execution takes at least minRunTime
		numCyclesGPU *= 2;
		startClock = clock();
		constraintSumGPU << <numBlocks, blockSize >> > (dataArray, arraySize, resultsGPU, numCyclesGPU, numConstraints, constraints, constraintSizes );
		hipDeviceSynchronize();
		endClock = clock();

		elapsedClocks = endClock - startClock;
		elapsedTimeGPU = ((double)(elapsedClocks)) / (CLOCKS_PER_SEC);
	}
	avgElapsedTimeGPU = elapsedTimeGPU / numCyclesGPU;
	printf("GPU: Elapsed time= %fs. Average execution time= %fs.\n\n", elapsedTimeGPU, avgElapsedTimeGPU);

	/* Print comparison results
	*
	*
	*
	*
	*/

	// free allocated memory
	genericFree(dataArray);
	genericFree(resultsCPU);
	genericFree(resultsGPU);
	// FREE CONSTRAINT SIZES AND CONSTRAINTS
	genericFree(constraintSizes);
	for (int i = 0; i < numConstraints; i++) {
		genericFree(constraints[i]);
	}
	genericFree(constraints);

	return 0;
}
