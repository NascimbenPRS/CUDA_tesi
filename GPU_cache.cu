#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <stdio.h>



// sum array of integers sequentially (using cache)
__global__ void arraySum(int *arr, int arraySize, int *sumValue) {
	int tempSum = 0;
		for (int i = 0; i < arraySize; i++) {
			tempSum += arr[i];
		}
	*sumValue = tempSum;
	printf("Sum value (thread): %d\n", *sumValue);
}

// kernel to initialize array
__global__ void initializeArray(int *arr, int arraySize) {
	for (int i = 0; i < arraySize; i++) {
		arr[i] = 1;
	}
	printf("Array initialized\n");
}

int main()
{

	int arraySize = 1 << 22; // 4M integers
	int numCycles = 10; // # of repetitions
	int *arr, *sumValue;

	printf("Sum array of integers on GPU (single thread) using cache.\nArray size=  %d integers\n", arraySize);
	

	hipMallocManaged(&arr, arraySize * sizeof(int)); // allocate arraySize * 4 bytes
	hipMallocManaged(&sumValue, sizeof(int));

	// initialize array

	/* initialize on CPU
	for (int i = 0; i < arraySize; i++) {
		arr[i] = 1;
	}

	*/
	

	initializeArray << <1, 1 >> > (arr, arraySize);
	hipDeviceSynchronize();

	*sumValue = 0;

	/*
	// Prefetch data to GPU
	int device = -1;
	hipGetDevice(&device); 
	hipDeviceSynchronize();
	hipMemPrefetchAsync(sumValue, sizeof(int), device, NULL);
	hipDeviceSynchronize();
	printf("Device used: %d\n", device);
	*/

	// Time measurement
	double elapsedTime;
	int elapsedClocks = 0, startClock = 0, endClock = 0;
	startClock = clock();

	for (int i= 0; i < numCycles; i++){
		arraySum << <1, 1 >> > (arr, arraySize, sumValue);
		hipDeviceSynchronize();
		//printf("Call number: %d\n", i + 1);
	}


	endClock = clock();

	// Print results 
	printf("startClock: %d\n", startClock);
	printf("endClock: %d\n", endClock);
	elapsedClocks = endClock - startClock;
	printf("elapsedClock: %d\n", elapsedClocks);
	elapsedTime = ((double)(elapsedClocks)) / (CLOCKS_PER_SEC * numCycles); // average execution time
	printf("Sum = %d, elapsed time= %f s.\n", *sumValue, elapsedTime);


	hipFree(arr);
	hipFree(sumValue);
	return 0;
}