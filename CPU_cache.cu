#include "hip/hip_runtime.h"
//#include "device_launch_parameters.h"

#include <iostream>
#include <stdio.h>


// sum array of integers sequentially (using cache)
void arraySum(int *arr, int arraySize, int *sumValue) {
	int tempSum = 0;
	for (int i = 0; i < arraySize; i++) {
		tempSum += arr[i];
	}
	*sumValue = tempSum;
}

int main()
{
	int arraySize = 1 << 22; // 4M integers
	int numCycles = 2000; // # of repetitions
	int *arr;
	hipMallocManaged(&arr, arraySize * sizeof(int)); // allocate arraySize * 4 bytes
	printf("Sum array of integers on CPU, using cache.\nArray size=  %d integers\n", arraySize);

	// initialize array
	for (int i = 0; i < arraySize; i++) {
		arr[i] = 1;
	}

	int sumValue = 0;
	int elapsedClocks = 0, startClock = 0, endClock = 0;
	double elapsedTime;
	
	// Time measurement
	startClock = clock();
	for (int j = 0; j < numCycles; j++) {
		arraySum(arr, arraySize, &sumValue);

	}

	endClock = clock();

	// Print results
	printf("startClock: %d\n", startClock);
	printf("endClock: %d\n", endClock);
	elapsedClocks = endClock - startClock;
	printf("elapsedClock: %d\n", elapsedClocks);
	elapsedTime = ((double)(elapsedClocks)) / (CLOCKS_PER_SEC * numCycles);
	printf("Sum = %d, elapsed time= %f s.\n", sumValue, elapsedTime);

	hipFree(arr);

	return 0;
}
