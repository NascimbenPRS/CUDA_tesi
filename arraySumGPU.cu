#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <stdio.h>



// sum array of integers sequentially (using cache)
__global__ void arraySumGPU(int *arr, int arraySize, int *sumValue, int numCycles) {
	int tempSum = 0;
	for (int k = 0; k < numCycles; k++) {
		tempSum = 0;
		for (int i = 0; i < arraySize; i++) {
			tempSum += arr[i];
		}
	}
	*sumValue = tempSum;
	//printf("Sum value (thread): %d\n", *sumValue);
}

// sum array of integers non-sequentially (not using cache)
__global__ void arraySumStrideGPU(int *arr, int arraySize, int *sumValue, int numCycles, int cacheLineSize) {
	int tempSum = 0;
	for (int k = 0; k < numCycles; k++) {
		tempSum = 0;
		for (int i = 0; i < cacheLineSize; i++) {
			for (int j = i; j < arraySize; j += cacheLineSize) {
				tempSum += arr[j];
			}
		}
	}
	*sumValue = tempSum;
}

// kernel to initialize array
__global__ void initializeArray(int *arr, int arraySize) {
	for (int i = 0; i < arraySize; i++) {
		arr[i] = 1;
	}
	printf("Array initialized\n");
}

// allocate memory, using either malloc or hipMallocManaged
void genericMalloc(void *ptr, int size, char *alloc_mode) {
	if (strcmp(alloc_mode, "GPU") == 0) {
		hipMallocManaged(&ptr, size);
		hipDeviceSynchronize();
		printf("allocate on gpu\n");
	}
	else {
		if (strcmp(alloc_mode, "CPU") == 0) {
			ptr = malloc(size);
		}
	}
}

// free memory, using either free or hipFree
void genericFree(void *ptr, char *alloc_mode) {
	if (strcmp(alloc_mode, "GPU") == 0) {
		hipFree(ptr);
		hipDeviceSynchronize();
	}
	else {
		if (strcmp(alloc_mode, "CPU") == 0) {
			free(ptr);
		}
	}
}

int main(int argc, char *argv[])
{
	int arraySize = 1 << 23; // 8M integers
	int usesCache = 1; // 0: don't use cache, 1: use cache (default)
	int cacheLineSize = 128 / sizeof(int); // # integers per cache line
	int numCycles = 40; // # of repetitions
	int *arr, *sumValue;

	hipMallocManaged(&arr, arraySize * sizeof(int)); // allocate arraySize * 4 bytes
	hipMallocManaged(&sumValue, sizeof(int));
	/*
	genericMalloc(arr, arraySize * sizeof(int), "GPU");
	genericMalloc(sumValue, sizeof(int), "GPU");
	*/

	printf("Sum array of integers on GPU (single thread). Array size=  %d integers\n", arraySize);


	// check for command line options
	if (argc < 2) {
		printf("No options specified, use cache by default. no_cache option available\n");
	}
	else {
		// options specified
		if (strcmp(argv[1], "no_cache") == 0) {
			usesCache = 0; 
			printf("no_cache option specified, don't use cache.\n");
		}
		else {
			printf("Unsupported option, use cache by default. no_cache option available\n");
		}
	}


	// initialize data

	// initialize array on CPU
	for (int i = 0; i < arraySize; i++) {
		arr[i] = 1;
	}
	*sumValue = 0;

	// Prefetch data to GPU
	int device = -1;
	hipGetDevice(&device);
	hipDeviceSynchronize();
	hipMemPrefetchAsync(arr, arraySize * sizeof(int), device, NULL);
	hipMemPrefetchAsync(sumValue, sizeof(int), device, NULL);
	hipDeviceSynchronize();
	//printf("Device used: %d\n", device);


	// Time measurement
	int elapsedClocks = 0, startClock = 0, endClock = 0;
	double elapsedTime, avgElapsedTime;

	startClock = clock();
	if (usesCache) {
		arraySumGPU << <1, 1 >> > (arr, arraySize, sumValue, numCycles);
	}
	else {
		arraySumStrideGPU << <1, 1 >> > (arr, arraySize, sumValue, numCycles, cacheLineSize);
	}
	hipDeviceSynchronize();
	endClock = clock();

	// Print results 
	printf("startClock: %d\n", startClock);
	printf("endClock: %d\n", endClock);
	elapsedClocks = endClock - startClock;
	printf("elapsedClock: %d\n", elapsedClocks);
	elapsedTime = ((double)(elapsedClocks)) / (CLOCKS_PER_SEC);
	avgElapsedTime = elapsedTime / numCycles;
	printf("Sum= %d. Number of repetitions= %d.\nElapsed time= %fs. Average elapsed time= %fs.\n1n", *sumValue, numCycles, elapsedTime, avgElapsedTime);

	
	hipFree(arr);
	hipFree(sumValue);
	

	/*
	genericFree(arr, "GPU");
	genericFree(sumValue, "GPU");
	*/
	hipDeviceReset();
	return 0;
}