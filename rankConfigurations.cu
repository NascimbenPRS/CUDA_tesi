#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
//#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>

#include "commonFunctions.h" // generic malloc
#include "sumOptions.h" // options handling


/*
How to compile:
	nvcc "filename" anyoption.cpp
*/


// sum array of integers sequentially (using cache)
void arraySum(int *arr, int arraySize, int *sumValue, int numCycles) {
	int tempSum = 0;
	for (int k = 0; k < numCycles; k++) {
		tempSum = 0;
		for (int i = 0; i < arraySize; i++) {
			tempSum += arr[i];
		}
	}
	*sumValue = tempSum;
}

// sum array of integers non-sequentially (not using cache)
void arraySumStride(int *arr, int arraySize, int *sumValue, int numCycles, int cacheLineSize) {
	int tempSum = 0;
	for (int k = 0; k < numCycles; k++) {
		tempSum = 0;
		for (int i = 0; i < cacheLineSize; i++) {
			for (int j = i; j < arraySize; j += cacheLineSize) {
				tempSum += arr[j];
			}
		}
	}
	*sumValue = tempSum;
}

// sum array on multiple threads on GPU
__global__ void arraySumGPU(int **arr, int arraySize, int *sumValue, int numCycles) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	int tempSum = 0;
	for (int k = 0; k < numCycles; k++) {
		tempSum = 0;
		for (int i = 0; i < arraySize; i++) {
			tempSum += arr[index][i];
		}
	}
	sumValue[index] = tempSum;
}

// sum array of integers non-sequentially (not using cache) on multiple threads on GPU
__global__ void arraySumStrideGPU(int **arr, int arraySize, int *sumValue, int numCycles, int cacheLineSize) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	int tempSum = 0;
	for (int k = 0; k < numCycles; k++) {
		tempSum = 0;
		for (int i = 0; i < cacheLineSize; i++) {
			for (int j = i; j < arraySize; j += cacheLineSize) {
				tempSum += arr[index][j];
			}
		}
	}
	sumValue[index] = tempSum;
}

// initialize array of arrays using multiple threads
__global__ void initializeArraysGPU(int **arrGPU, int arraySize) {
	int index = threadIdx.x; // current thread ID
	for (int i = 0; i < arraySize; i++) {
		arrGPU[index][i] = 1;
	}
}


// struct to keep track of performances
struct performance
{
	int numThreads= 0;
	int numBlocks= 0;
	int numThreadsTotal= 0;
	double timeCPU= 0.f;
	double timeGPU= 0.f;
	double speedup= 0.f;
};


void printBestPerformances(struct performance *bestPerf, int numSavedPerformances) {
	printf("\nBest %d performances:\n", numSavedPerformances);
	for (int i = 0; i < numSavedPerformances; i++) {
		printf("%d) %d blocks, %d threads per block, %d threads running\n", i+1, bestPerf[i].numBlocks, bestPerf[i].numThreads, bestPerf[i].numThreadsTotal);
		printf("-- CPU time: %f, GPU time: %f, speedup= %f\n", bestPerf[i].timeCPU, bestPerf[i].timeGPU, bestPerf[i].speedup);
	}
	printf("-------------\n");
}

void updateBestPerformances(struct performance *bestPerf, int numSavedPerformances, struct performance perf) {
	int i = 0;
	while (i < numSavedPerformances) {
		if (perf.speedup > bestPerf[i].speedup) {
			// shift performances
			for (int k = numSavedPerformances - 1; k > i; k--) {
				bestPerf[k] = bestPerf[k - 1];
			}
			bestPerf[i] = perf;
			i = numSavedPerformances; // force exit
		}
		else {
			i++;
		}
	}
}

// print the best performance running searchedNumThreads concurrent threads in bestPerf
void printBestPerformanceNumThreads(struct performance *bestPerf, int numSavedPerformances, int searchedNumThreads) {
	int i = 0;
	while (i < numSavedPerformances) {
		if (bestPerf[i].numThreadsTotal == searchedNumThreads) {
			printf("Running threads: %d\n", searchedNumThreads);
			printf("-- %d blocks, %d threads per block\n", bestPerf[i].numBlocks, bestPerf[i].numThreads);
			printf("-- CPU time: %f, GPU time: %f, speedup= %f\n\n", bestPerf[i].timeCPU, bestPerf[i].timeGPU, bestPerf[i].speedup);
			i = numSavedPerformances; // force exit
		}
		else {
			i++;
		}
	}
}




/*
Ranks all <<< gridSize, blockSize>>> GPU kernel configurations, where:
-- gridSize <= maxNumBlocks;
-- (gridSize x blocksize) <= maxNumThreads.

Configurations are ranked by speedup (i.e. CPU_Time/GPU_Time), where:
-- CPU_Time= time to sum numThreadsTotal arrays of integers on CPU (serially);
-- GPU_Time= time to sum numThreadsTotal array of integers on GPU, using a certain kernel configuration.


Shows the best configuration for each possible number of threads (1, 2, .., maxNumThreads)
*/

int main(int argc, char *argv[])
{
	int arraySize = 1 << 20; // 1M integers
	int usesCache = 1; // 0: don't use cache, 1: use cache (default)
	int cacheLineSize = 64 / sizeof(int); // # integers per cache line on CPU
	int cacheLineSizeGPU = 128 / sizeof(int); // # integers per cache line on GPU
	int numCycles = 1000; // default # of repetitions on CPU
	int numCyclesGPU = 30; // default # of repetitions on GPU
	int *arrCPU, *sumValueCPU, **arrGPU, *sumValuesGPU; // arrGPU: array of arrays
	int numThreads = 1; // # of threads per block
	int numBlocks = 1; // # of blocks
	int numThreadsTotal; // # of threads running concurrently (= numThreads * numBlocks)
	int maxNumThreads = 1024; // max # of threads running concurrently
	int maxNumBlocks = 512; // max # of usable blocks
	
	// create an array to save performances
	int numSavedPerformances = 200; // # best performances to record
	struct performance *bestPerformances;
	genericMalloc((void**)&bestPerformances, numSavedPerformances * sizeof(struct performance));



	hipDeviceProp_t prop;
	int device = -1;
	hipGetDevice(&device);
	hipDeviceSynchronize();
	hipGetDeviceProperties(&prop, device);
	int numSM = prop.multiProcessorCount;

	printf("\nCompare execution time on CPU and GPU\n");
	printf("Default options: use cache, arraySize= %d integers.\n", arraySize);
	//printf("-- Default number of repetitions: %d (CPU), %d (GPU).\n", numCycles, numCyclesGPU);
	printf("-- number of multiprocessors on GPU= %d\n", numSM);
	printf("-- max number of blocks= %d.\n", maxNumBlocks);
	printf("-- max number of threads= %d.\n", maxNumThreads);


	readOptions(argc, argv, &usesCache, &numCycles, &numCyclesGPU, &arraySize); // read options from command line and update values accordingly
	// don't use --rep


	// allocate memory for CPU execution
	genericMalloc((void**)&arrCPU, arraySize * sizeof(int));
	genericMalloc((void**)&sumValueCPU, sizeof(int));

	// initialize array on CPU
	for (int i = 0; i < arraySize; i++) {
		arrCPU[i] = 1;
	}
	*sumValueCPU = 0;


	// Time measurement
	int minRunTime = 5; // elapsedTime must be at least minRunTime seconds
	int minNumCyclesCPU = 1, minNumCyclesGPU = 1;
	int elapsedClocks = 0, startClock = 0, endClock = 0;
	double elapsedTimeCPU = 0.f, avgElapsedTimeCPU = 0.f, elapsedTimeGPU = 0.f, avgElapsedTimeGPU = 0.f;
	double speedup = 0.f; // CPU time/GPU time


	while (numBlocks <= maxNumBlocks) {
		numThreadsTotal = numBlocks * numThreads;
		printf("NUM BLOCKS= %d, NUM THREADS PER BLOCK = %d, TOTAL NUM THREADS= %d\n", numBlocks, numThreads, numThreadsTotal);
		// Measure CPU execution time
		elapsedTimeCPU = 0.f;
		avgElapsedTimeCPU = 0.f;
		numCycles = minNumCyclesCPU;
		while (elapsedTimeCPU < minRunTime) {// double numCycles until execution takes at least minRunTime
			numCycles *= 2;
			startClock = clock();
			if (usesCache) {
				arraySum(arrCPU, arraySize, sumValueCPU, numThreadsTotal * numCycles);
			}
			else {
				arraySumStride(arrCPU, arraySize, sumValueCPU, numThreadsTotal * numCycles, cacheLineSize);
			}
			endClock = clock();

			elapsedClocks = endClock - startClock;
			elapsedTimeCPU = ((double)(elapsedClocks)) / (CLOCKS_PER_SEC);
		}
		avgElapsedTimeCPU = elapsedTimeCPU / numCycles; // = avg time * numThreadsTotal
		//printf("--CPU: Elapsed time= %fs. \n", elapsedTimeCPU);
		printf("-- CPU: Average execution time= %fs.\n", avgElapsedTimeCPU);


		// Measure GPU execution time

		// allocate memory for GPU execution
		genericMalloc((void**)&arrGPU, numThreadsTotal * sizeof(int*));
		genericMalloc((void**)&sumValuesGPU, numThreadsTotal * sizeof(int));
		// allocate for each array copy
		for (int i = 0; i < numThreadsTotal; i++) {
			genericMalloc((void**)&arrGPU[i], arraySize * sizeof(int));
		}
		// initialize arrays on GPU
		initializeArraysGPU << <1, numThreadsTotal >> > (arrGPU, arraySize);
		hipDeviceSynchronize();

		elapsedTimeGPU = 0.f;
		avgElapsedTimeGPU = 0.f;
		numCyclesGPU = minNumCyclesGPU;
		while (elapsedTimeGPU < minRunTime) {// double numCyclesGPU until execution takes at least minRunTime
			numCyclesGPU *= 2;
			startClock = clock();
			if (usesCache) {
				arraySumGPU << <numBlocks, numThreads >> > (arrGPU, arraySize, sumValuesGPU, numCyclesGPU);
			}
			else {
				arraySumStrideGPU << <numBlocks, numThreads >> > (arrGPU, arraySize, sumValuesGPU, numCyclesGPU, cacheLineSizeGPU);
			}
			hipDeviceSynchronize();
			endClock = clock();

			elapsedClocks = endClock - startClock;
			elapsedTimeGPU = ((double)(elapsedClocks)) / (CLOCKS_PER_SEC);
		}

		avgElapsedTimeGPU = elapsedTimeGPU / numCyclesGPU;
		speedup = avgElapsedTimeCPU / avgElapsedTimeGPU;
		// printf("--GPU: Elapsed time= %fs. \n", elapsedTimeGPU);
		printf("-- GPU: Average execution time= %fs.\n", avgElapsedTimeGPU);
		printf("--speedup : %f\n\n", speedup);

		// update record of best performances
		struct performance current_perf = { numThreads, numBlocks, numThreadsTotal, avgElapsedTimeCPU, avgElapsedTimeGPU, speedup };
		updateBestPerformances(bestPerformances, numSavedPerformances, current_perf);

		// Free GPU memory
		genericFree(sumValuesGPU);
		for (int i = 0; i < numThreadsTotal; i++) {
			genericFree(arrGPU[i]);
		}
		genericFree(arrGPU);

		if (numThreadsTotal >= maxNumThreads) {
			// double numBlocks and reset numThreads
			numBlocks *= 2;
			numThreads = 1;
			printf("-------------------\n\n");
		}
		else {
			numThreads *= 2;
		}
	}


	// Print overall results
	printBestPerformances(bestPerformances, numSavedPerformances);
	// Print best performance for each possible number of threads
	for (int i = 1; i <= maxNumThreads; i *= 2) {
		printBestPerformanceNumThreads(bestPerformances, numSavedPerformances, i);
	}
	genericFree(bestPerformances);
	

	// free allocated memory for CPU
	genericFree(arrCPU);
	genericFree(sumValueCPU);

	

	return 0;
}
