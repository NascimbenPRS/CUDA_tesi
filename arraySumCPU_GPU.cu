#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
//#include ""

#include <iostream>
#include <stdio.h>
#include <string.h>

#include "commonFunctions.h" // generic malloc
#include "anyoption.h" // options parsing

/*
How to compile:
	nvcc/gcc/... "filename" anyoption.cpp
*/


// sum array of integers sequentially (using cache)
void arraySum(int *arr, int arraySize, int *sumValue, int numCycles) {
	int tempSum = 0;
	for (int k = 0; k < numCycles; k++) {
		tempSum = 0;
		for (int i = 0; i < arraySize; i++) {
			tempSum += arr[i];
		}
	}
	*sumValue = tempSum;
}

// sum array of integers non-sequentially (not using cache)
void arraySumStride(int *arr, int arraySize, int *sumValue, int numCycles, int cacheLineSize) {
	int tempSum = 0;
	for (int k = 0; k < numCycles; k++) {
		tempSum = 0;
		for (int i = 0; i < cacheLineSize; i++) {
			for (int j = i; j < arraySize; j += cacheLineSize) {
				tempSum += arr[j];
			}
		}
	}
	*sumValue = tempSum;
}

// sum array of integers sequentially (using cache) on GPU
__global__ void arraySumGPU(int *arr, int arraySize, int *sumValue, int numCycles) {
	int tempSum = 0;
	for (int k = 0; k < numCycles; k++) {
		tempSum = 0;
		for (int i = 0; i < arraySize; i++) {
			tempSum += arr[i];
		}
	}
	*sumValue = tempSum;
	//printf("Sum value (thread): %d\n", *sumValue);
}

// sum array of integers non-sequentially (not using cache) on GPU
__global__ void arraySumStrideGPU(int *arr, int arraySize, int *sumValue, int numCycles, int cacheLineSize) {
	int tempSum = 0;
	for (int k = 0; k < numCycles; k++) {
		tempSum = 0;
		for (int i = 0; i < cacheLineSize; i++) {
			for (int j = i; j < arraySize; j += cacheLineSize) {
				tempSum += arr[j];
			}
		}
	}
	*sumValue = tempSum;
}


int main(int argc, char *argv[])
{
	int arraySize = 1 << 23; // 8M integers
	int usesCache = 1; // 0: don't use cache, 1: use cache (default)
	int cacheLineSize = 64 / sizeof(int); // # integers per cache line on CPU
	int cacheLineSizeGPU = 128 / sizeof(int); // # integers per cache line on GPU
	int numCycles = 1000; // default # of repetitions on CPU
	int numCyclesGPU = 30; // default # of repetitions on GPU
	int *arr, *sumValue;
	printf("Default options: use cache, arraySize= %d integers.\n", arraySize);
	printf("-- Default number of repetitions: %d (CPU), %d (GPU).\n", numCycles, numCyclesGPU);


	// parse options
	AnyOption *opt = new AnyOption();
	// set usage
	opt->addUsage("Options usage: ");
	opt->addUsage("");
	opt->addUsage(" --no_cache \tDon't use cache ");
	opt->addUsage(" --rep <rep>\tNumber of repetitions ");
	opt->addUsage(" --size <size>\tArray size (* 2^20) elements");
	opt->addUsage("");
	opt->printUsage();

	// set options
	opt->setFlag("no_cache");
	opt->setOption("rep");
	opt->setOption("size");

	// Process commandline and get the options
	opt->processCommandArgs(argc, argv);

	// Get option values

	if (opt->getFlag("no_cache")) {
		usesCache = 0;
		printf("no_cache flag set\n");
	}
	if (opt->getValue("rep") != NULL) {
		numCycles = atoi(opt->getValue("rep"));
		numCyclesGPU = numCycles;
		printf("Number of repetitions set to: %d\n", numCycles);
	}
	if (opt->getValue("size") != NULL) {
		arraySize = (1 << 20) * atoi(opt->getValue("size"));
		printf("Array size set to: %dM integers\n", arraySize);
	}

	delete opt;
	// options parsed

	// allocate memory
	genericMalloc((void**)&arr, arraySize * sizeof(int));
	genericMalloc((void**)&sumValue, arraySize * sizeof(int));

	// initialize array on CPU
	for (int i = 0; i < arraySize; i++) {
		arr[i] = 1;
	}
	*sumValue = 0;


	// Time measurement
	int elapsedClocks = 0, startClock = 0, endClock = 0;
	double elapsedTime, avgElapsedTime;


#ifndef __NVCC__
	printf("Not compiled with NVCC, run on CPU\n");
	startClock = clock();
	if (usesCache) {
		arraySum(arr, arraySize, sumValue, numCycles);
	}
	else {
		arraySumStride(arr, arraySize, sumValue, numCycles, cacheLineSize);
	}
#endif
#ifdef __NVCC__
	printf("Compiled with NVCC, run on GPU\n");
	// Prefetch data to GPU
	int device = -1;
	hipGetDevice(&device);
	hipDeviceSynchronize();
	hipMemPrefetchAsync(arr, arraySize * sizeof(int), device, NULL);
	hipMemPrefetchAsync(sumValue, sizeof(int), device, NULL);
	hipDeviceSynchronize();

	startClock = clock();
	if (usesCache) {
		arraySumGPU << <1, 1 >> > (arr, arraySize, sumValue, numCyclesGPU);
	}
	else {
		arraySumStrideGPU << <1, 1 >> > (arr, arraySize, sumValue, numCyclesGPU, cacheLineSizeGPU);
	}
	hipDeviceSynchronize();
#endif
	endClock = clock();


	// Print results
	printf("startClock: %d, endClock: %d\n", startClock, endClock);
	elapsedClocks = endClock - startClock;
	printf("elapsedClock: %d\n", elapsedClocks);
	elapsedTime = ((double)(elapsedClocks)) / (CLOCKS_PER_SEC);
#ifndef __NVCC__
	avgElapsedTime = elapsedTime / numCycles;
#endif
#ifdef __NVCC__
	avgElapsedTime = elapsedTime / numCyclesGPU;
#endif
	printf("Sum= %d. \nElapsed time= %fs. Average execution time= %fs.\n\n", *sumValue, elapsedTime, avgElapsedTime);


	// free allocated memory
	genericFree(arr);
	genericFree(sumValue);

	return 0;
}
