#include "hip/hip_runtime.h"
//#include "device_launch_parameters.h"

#include <iostream>
#include <stdio.h>
#include <string.h>


// sum array of integers sequentially (using cache)
void arraySum(int *arr, int arraySize, int *sumValue, int numCycles) {
	int tempSum = 0;
	for (int k = 0; k < numCycles; k++) {
		tempSum = 0;
		for (int i = 0; i < arraySize; i++) {
			tempSum += arr[i];
		}
	}
	*sumValue = tempSum;
}

// sum array of integers non-sequentially (not using cache)
void arraySumStride(int *arr, int arraySize, int *sumValue, int numCycles, int cacheLineSize) {
	int tempSum = 0;
	for (int k = 0; k < numCycles; k++) {
		tempSum = 0;
		for (int i = 0; i < cacheLineSize; i++) {
			for (int j = i; j < arraySize; j += cacheLineSize) {
				tempSum += arr[j];
			}
		}
	}	
	*sumValue = tempSum;
}

// allocate memory, using either malloc or cudaMallocManaged
void genericMalloc(void *ptr, int size, char *alloc_mode) {
	if (strcmp(alloc_mode, "GPU") == 0) {
		hipMallocManaged(&ptr, size);
		hipDeviceSynchronize();
		printf("allocate on gpu\n");
	}
	else {
		if (strcmp(alloc_mode, "CPU") == 0) {
			ptr = malloc(size);
		}
	}
}

// free memory, using either free or cudaFree
void genericFree(void *ptr, char *alloc_mode){
	if (strcmp(alloc_mode, "GPU") == 0) {
		hipFree(ptr);
		hipDeviceSynchronize();
	}
	else {
		if (strcmp(alloc_mode, "CPU") == 0) {
			free(ptr);
		}
	}
}

int main(int argc, char *argv[])
{
	int arraySize = 1 << 23; // 8M integers
	int usesCache = 1; // 0: don't use cache, 1: use cache (default)
	int cacheLineSize = 64 / sizeof(int); // # integers per cache line
	int numCycles = 1000; // # of repetitions
	int *arr;
	arr = (int*) malloc(arraySize * sizeof(int));
	//cudaMallocManaged(&arr, arraySize * sizeof(int)); // allocate arraySize * 4 bytes
	//genericMalloc(arr, arraySize * sizeof(int), "CPU");
	printf("Sum array of integers on CPU. Array size=  %d integers\n", arraySize);


	// check for command line options
	if (argc < 2) {
		printf("No options specified, use cache by default. no_cache option available\n");
	}
	else {
		// options specified
		if (strcmp(argv[1], "no_cache") == 0) {
			usesCache = 0;
			printf("no_cache option specified, don't use cache.\n");
		}
		else {
			printf("Unsupported option, use cache by default. no_cache option available\n");
		}
	}


	// initialize array
	for (int i = 0; i < arraySize; i++) {
		arr[i] = 1;
	}
	int sumValue = 0;
	

	// Time measurement
	int elapsedClocks = 0, startClock = 0, endClock = 0;
	double elapsedTime, avgElapsedTime;

	startClock = clock();
	if (usesCache) {
		arraySum(arr, arraySize, &sumValue, numCycles);
	}
	else {
		arraySumStride(arr, arraySize, &sumValue, numCycles, cacheLineSize);
	}
	endClock = clock();


	// Print results
	printf("startClock: %d\n", startClock);
	printf("endClock: %d\n", endClock);
	elapsedClocks = endClock - startClock;
	printf("elapsedClock: %d\n", elapsedClocks);
	elapsedTime = ((double)(elapsedClocks)) / (CLOCKS_PER_SEC);
	avgElapsedTime = elapsedTime / numCycles;
	printf("Sum= %d. Number of repetitions= %d.\nElapsed time= %fs. Average elapsed time= %fs.\n\n", sumValue, numCycles, elapsedTime, avgElapsedTime);

	free(arr);
	//genericFree(arr, "CPU");

	return 0;
}
